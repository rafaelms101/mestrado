#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "mycuda.h"
#include "topk.cu"

#include "hip/hip_runtime_api.h"

#include <cstdio>

#define safe_call(call) if (hipSuccess != call) { err = hipGetLastError(); \
													fprintf(stderr, "Failed call: %s\nError: %s\n", \
															#call, hipGetErrorString(err)); \
													exit(EXIT_FAILURE); }


extern __shared__ char shared_memory[];

#define ACTIVE_BLOCKS 10


//TODO: make the merge of the w query results in the GPU (?)
__global__ void compute_dists(pqtipo PQ, mat residual, ivf_t* ivf,
		int* entry_map, int* starting_imgid, int* starting_inputid,
		Img* full_input, matI idxs, mat dists, int k) {
	int tid = threadIdx.x;
	int nthreads = blockDim.x;
	int bid = blockIdx.x;
	int numBlocks = gridDim.x;

	float* distab = (float*) shared_memory;

	for (int qid = bid; qid < residual.n; qid += numBlocks) {
		//computing disttab
		float* current_residual = residual.mat + qid * PQ.nsq * PQ.ds;
		int step_size = (PQ.ks * PQ.nsq + nthreads - 1) / nthreads;

		int begin_i = tid * step_size;
		int end_i = min(begin_i + step_size, PQ.ks * PQ.nsq) - 1;
		float* centroid = PQ.centroids + begin_i * PQ.ds;

		for (int i = begin_i; i <= end_i; i++) {
			int d = i / PQ.ks;

			float* sub_residual = current_residual + d * PQ.ds;
			float dist = 0;

			for (int j = 0; j < PQ.ds; j++, centroid++) {
				float diff = sub_residual[j] - *centroid;
				dist += diff * diff;
			}

			distab[i] = dist;
		}

		__syncthreads();

		//computing the distances to the vectors
		ivf_t entry = ivf[entry_map[qid]];
		Img* input = full_input + starting_inputid[qid];

		for (int i = tid; i < entry.idstam; i += nthreads) {
			float dist = 0;

			for (int s = 0; s < PQ.nsq; s++) {
				dist += distab[PQ.ks * s + entry.codes.mat[PQ.nsq * i + s]];
			}

			input[i] = {dist, entry.ids[i]};
		}

		__syncthreads();
		//choosing the top k


		//TODO: remember to analyze the case where size < k or size < 2k
		// selecting num_heaps
		auto shared_memory_size = 48 << 10; //TODO: there might be some function to obtain the shared memory size from the environment
		auto heap_size = k * sizeof(Entry<Img>);
		auto max_heaps = shared_memory_size / heap_size;
		auto num_subheaps = max_heaps - 1;

		if (num_subheaps > blockDim.x) num_subheaps = blockDim.x;

		if (num_subheaps * 2 * k > entry.idstam) {
			num_subheaps = entry.idstam / (2 * k);
		}

		if (num_subheaps == 0) num_subheaps = 1;

		topk(qid, num_subheaps, k, full_input, starting_inputid, dists.mat,
				idxs.mat);

		__syncthreads();
	}
}

hipError_t alloc(void **devPtr, size_t size) {
	return hipMalloc(devPtr, size);
}



void core_gpu(pqtipo PQ, mat residual, ivf_t* ivf, int ivf_size, int* entry_map, int* starting_imgid,  int* starting_inputid,  int num_imgs, matI idxs, mat dists, int k) {
//	int numBlocks;
//	hipOccupancyMaxActiveBlocksPerMultiprocessor (&numBlocks, compute_dists, 1024, 48 << 10);
//	std::printf("Maximum number of blocks: %d\n", numBlocks);


	//TODO: implement / redo the error handling so that we have less code duplication
	hipError_t err = hipSuccess;

	pqtipo gpu_PQ = PQ;

	std::printf("Allocating %d MB for centroids\n",  sizeof(float) * PQ.centroidsd * PQ.centroidsn / 1024 / 1024);
	safe_call(alloc((void **) &gpu_PQ.centroids, sizeof(float) * PQ.centroidsd * PQ.centroidsn));
	safe_call(hipMemcpy(gpu_PQ.centroids, PQ.centroids, sizeof(float) * PQ.centroidsd * PQ.centroidsn, hipMemcpyHostToDevice));

	mat gpu_residual = residual;
	std::printf("Allocating %d MB for residuals\n",  sizeof(float) * residual.n * residual.d / 1024 / 1024);
	safe_call(alloc((void **) &gpu_residual.mat, sizeof(float) * residual.n * residual.d));
	safe_call(hipMemcpy(gpu_residual.mat, residual.mat, sizeof(float) * residual.n * residual.d, hipMemcpyHostToDevice));


	long ivf_mem_size = 0;
	ivf_t* gpu_ivf;

	ivf_mem_size += sizeof(ivf_t) * ivf_size;
	std::printf("IVF memory size up to now: %d MB\n",  ivf_mem_size / 1024 / 1024);
	safe_call(alloc((void **) &gpu_ivf, sizeof(ivf_t) * ivf_size));


	ivf_t* tmp_ivf = new ivf_t[ivf_size];

	for (int i = 0; i < ivf_size; i++) {
		tmp_ivf[i].idstam = ivf[i].idstam;
		tmp_ivf[i].codes = ivf[i].codes;

		ivf_mem_size += sizeof(int) * tmp_ivf[i].idstam;
		std::printf("IVF memory size up to now: %d MB\n",  ivf_mem_size / 1024 / 1024);
		safe_call(alloc((void **) &tmp_ivf[i].ids, sizeof(int) * tmp_ivf[i].idstam));
		safe_call(hipMemcpy(tmp_ivf[i].ids, ivf[i].ids, sizeof(int) * ivf[i].idstam, hipMemcpyHostToDevice));

		ivf_mem_size += sizeof(int) * tmp_ivf[i].codes.n * tmp_ivf[i].codes.d;
		std::printf("IVF memory size up to now: %d MB\n",  ivf_mem_size / 1024 / 1024);
		safe_call(alloc((void **) &tmp_ivf[i].codes.mat, sizeof(int) * tmp_ivf[i].codes.n * tmp_ivf[i].codes.d));
		std::printf("entry=%d, idstam=%d, codes.n=%d, codes.d=%d\n", i, tmp_ivf[i].idstam, tmp_ivf[i].codes.n, tmp_ivf[i].codes.d);
		safe_call(hipMemcpy(tmp_ivf[i].codes.mat, ivf[i].codes.mat, sizeof(int) * ivf[i].codes.n * ivf[i].codes.d, hipMemcpyHostToDevice));
	}

	std::printf("Allocating %d MB for IVF\n",  ivf_mem_size / 1024 / 1024);
	safe_call(hipMemcpy(gpu_ivf, tmp_ivf, sizeof(ivf_t) * ivf_size, hipMemcpyHostToDevice));

	int* gpu_entry_map;
	std::printf("Allocating %d MB for entry map\n",  sizeof(int) * residual.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_entry_map, sizeof(int) * residual.n));
	safe_call(hipMemcpy(gpu_entry_map, entry_map, sizeof(int) * residual.n, hipMemcpyHostToDevice));

	int* gpu_starting_imgid;
	std::printf("Allocating %d MB for starting img id\n",  sizeof(int) * residual.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_starting_imgid, sizeof(int) * residual.n));
	safe_call(hipMemcpy(gpu_starting_imgid, starting_imgid, sizeof(int) * residual.n, hipMemcpyHostToDevice));

	//query_id_t* gpu_elements;
	//safe_call(hipMalloc((void **) &gpu_elements, sizeof(query_id_t) * residual.n)); //TODO: I dont know if this is truly needed
	//safe_call(hipMemcpy(gpu_elements, elements, sizeof(query_id_t) * residual.n, hipMemcpyHostToDevice));// TODO: need to rethink this

	matI gpu_idxs = idxs;

	std::printf("Allocating %d MB for idxs\n", sizeof(int) * idxs.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_idxs.mat, sizeof(int) * idxs.n));

	mat gpu_dists = dists;
	std::printf("Allocating %d MB for dists\n", sizeof(float) * dists.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_dists.mat, sizeof(float) * dists.n));

	//allocating the input buffer
	int* gpu_starting_inputid;
	std::printf("Allocating %d MB for input buffer\n", sizeof(int) * (residual.n + 1) / 1024 / 1024);
	safe_call(alloc((void **) &gpu_starting_inputid, sizeof(int) * (residual.n + 1)));
	safe_call(hipMemcpy(gpu_starting_inputid, starting_inputid, sizeof(int) * (residual.n + 1), hipMemcpyHostToDevice));

	Img* gpu_input;
	std::printf("Number of images: %d\n", num_imgs);
	std::printf("Image: %d\n", sizeof(Img));
	std::printf("Allocating %d MB for images\n",  sizeof(Img) * num_imgs / 1024 / 1024);
	safe_call(alloc((void **) &gpu_input, sizeof(Img) * num_imgs));

	dim3 block(1024, 1, 1);
	dim3 grid(ACTIVE_BLOCKS, 1, 1);

	//find biggest ivf entry
	int biggest = 0;
	for (int i = 0; i < ivf_size; i++ ) if (ivf[i].idstam > biggest) biggest = ivf[i].idstam;

	int sm_size = 48 << 10;

	std::printf("Trying to allocate: %dKB in shared memory\n", 48 << 10 / 1024);

	compute_dists<<<grid, block,  sm_size>>>(gpu_PQ, gpu_residual, gpu_ivf, gpu_entry_map, gpu_starting_imgid, gpu_starting_inputid, gpu_input, gpu_idxs, gpu_dists, k);

	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch compute_dists kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else std::printf("SUCESSS!\n");

	std::printf("After calling the kernel\n");

	//exit(0);

	//RECEIVING DATA FROM GPU
	//safe_call(hipMemcpy(elements, 0, sizeof(query_id_t) * residual.n, hipMemcpyDeviceToHost));

	safe_call(hipMemcpy(idxs.mat, gpu_idxs.mat , sizeof(int) * idxs.n, hipMemcpyDeviceToHost));
	safe_call(hipMemcpy(dists.mat, gpu_dists.mat, sizeof(float) * dists.n, hipMemcpyDeviceToHost));

	//FREEING MEMORY
	hipFree(gpu_PQ.centroids);
	hipFree(gpu_residual.mat);
	hipFree(gpu_ivf);

	for (int i = 0; i < ivf_size; i++) {
		hipFree(tmp_ivf[i].ids);
		hipFree(tmp_ivf[i].codes.mat);
	}

	hipFree(gpu_entry_map);
	hipFree(gpu_starting_imgid);
	hipFree(gpu_idxs.mat);
	hipFree(gpu_dists.mat);
	hipFree(gpu_starting_inputid);
	hipFree(gpu_input);

	delete[] tmp_ivf;

}
