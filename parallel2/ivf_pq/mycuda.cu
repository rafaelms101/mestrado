#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "mycuda.h"
#include "topk.cu"

#include "hip/hip_runtime_api.h"

#include <cstdio>

#define safe_call(call) if (hipSuccess != call) { err = hipGetLastError(); \
													fprintf(stderr, "Failed call: %s (error code %s)!\n", \
															#call, hipGetErrorString(err)); \
													exit(EXIT_FAILURE); }


//TODO: remember to not execute queries that dont correspond to an entry on the problem

extern __shared__ char shared_memory[];

// PQ.ks * PQ.nsq must be a multiple of 1024
__global__ void compute_dists(pqtipo PQ, mat residual, ivf_t* ivf, int* entry_map, int* starting_imgid, int* starting_inputid, Img* original_input, matI idxs, mat dists, int best_k) {
	if (blockIdx.x == 0 && threadIdx.x == 0) {
			std::printf("IN_GPU\n");
		}
	
	int tid = threadIdx.x;
	int nthreads = blockDim.x;
	int qid = blockIdx.x;
	
	float* distab = (float*) shared_memory;
	
	//computing disttab
	int step = PQ.ks * PQ.nsq / nthreads; 
	int j = step * tid;
	int initial_d = j / PQ.ks;
	int initial_k = j % PQ.ks;
	
	int nd = max(step / PQ.ks, 1); //nd = 1
	int slice = min(step, PQ.ks); //slice = 2 
	
	//std::printf("step=%d, j=%d, initial_d=%d, initial_k=%d\n", step, j, initial_d, initial_k);
		
	float* current_residual = residual.mat + qid * PQ.nsq * PQ.ds;
	
	
	//std::printf("BEFORE THE BIG FOR\n");
	
	for (int d = initial_d; d < initial_d + nd; d++) {
		float* sub_residual = current_residual + d * PQ.ds;
		
		for (int k = initial_k; k < initial_k + slice; k++) {
			float* centroid = PQ.centroids + (d * PQ.ks + k) * PQ.ds;
			float dist = 0;

			for (int i = 0; i < PQ.ds; i++) {
				float diff = sub_residual[i] - centroid[i];
				dist += diff * diff;
			}
			
			distab[PQ.ks * d + k] = dist;
			//std::printf("PQ.ks=%d, d=%d, k=%d, distab[%d] = %f\n", PQ.ks, d, k, PQ.ks * d + k, dist);
		}
		
		initial_k = 0;
	}
	
	__syncthreads();
	
	//computing the distances to the vectors
	ivf_t entry = ivf[entry_map[qid]];
	Img* input = original_input + starting_inputid[qid];
	
	int block_size = blockDim.x;

	for (int i = tid; i < entry.idstam; i += block_size) {
		float dist = 0;

		for (int s = 0; s < PQ.nsq; s++) {
			dist += distab[PQ.ks * s + entry.codes.mat[PQ.nsq * i + s]];
		}
		
		input[i] = { dist, entry.ids[i] };
		//std::printf("input[%d] = %f\n", i, dist);
	}
	
	__syncthreads();

	// now selecting num_shards
	auto shared_memory_size = (48 << 10) - PQ.ks * PQ.nsq * sizeof(float);  // 48 KB
	//std::printf("SHARED MEMORY SIZE: %dKB\n",shared_memory_size / 1024);
	auto heap_size = best_k * sizeof(Entry<Img>);
	// shared_memory_size = (num_shards + 1) * heap_size <=>
	int num_shards = shared_memory_size / heap_size - 1;
	if (num_shards <= 0) {
		num_shards = 1;
	}
	auto shard_size = entry.idstam / num_shards;
	auto min_shard_size = 2 * best_k;
	if (shard_size < min_shard_size) {
		num_shards = entry.idstam / min_shard_size;
	}
	if (num_shards <= 0) {
		num_shards = 1;
	} else if (num_shards > 1024) {
		num_shards = 1024;
	}

	topk(num_shards, best_k, original_input, starting_inputid, dists.mat, idxs.mat);
	__syncthreads();
	
	if (blockIdx.x == 0 && threadIdx.x == 0) {
		std::printf("OUT_GPU\n");
	}
}

hipError_t alloc(void **devPtr, size_t size) {
	return hipMalloc(devPtr, size);
}



void core_gpu(pqtipo PQ, mat residual, ivf_t* ivf, int ivf_size, int* entry_map, int* starting_imgid,  int* starting_inputid,  int num_imgs, matI idxs, mat dists, int k) {
	//TODO: implement / redo the error handling so that we have less code duplication
	hipError_t err = hipSuccess;
	
	pqtipo gpu_PQ = PQ;

	std::printf("Allocating %d MB for centroids\n",  sizeof(float) * PQ.centroidsd * PQ.centroidsn / 1024 / 1024);
	safe_call(alloc((void **) &gpu_PQ.centroids, sizeof(float) * PQ.centroidsd * PQ.centroidsn));
	safe_call(hipMemcpy(gpu_PQ.centroids, PQ.centroids, sizeof(float) * PQ.centroidsd * PQ.centroidsn, hipMemcpyHostToDevice));
	
	mat gpu_residual = residual;
	std::printf("Allocating %d MB for residuals\n",  sizeof(float) * residual.n * residual.d / 1024 / 1024);
	safe_call(alloc((void **) &gpu_residual.mat, sizeof(float) * residual.n * residual.d));
	safe_call(hipMemcpy(gpu_residual.mat, residual.mat, sizeof(float) * residual.n * residual.d, hipMemcpyHostToDevice));
	
	
	long ivf_mem_size = 0;
	ivf_t* gpu_ivf;

	ivf_mem_size += sizeof(ivf_t) * ivf_size;
	std::printf("IVF memory size up to now: %d MB\n",  ivf_mem_size / 1024 / 1024);
	safe_call(alloc((void **) &gpu_ivf, sizeof(ivf_t) * ivf_size));
	

	ivf_t* tmp_ivf = new ivf_t[ivf_size];
	
	for (int i = 0; i < ivf_size; i++) {
		tmp_ivf[i].idstam = ivf[i].idstam; 
		tmp_ivf[i].codes = ivf[i].codes;
		
		ivf_mem_size += sizeof(int) * tmp_ivf[i].idstam;
		std::printf("IVF memory size up to now: %d MB\n",  ivf_mem_size / 1024 / 1024);
		safe_call(alloc((void **) &tmp_ivf[i].ids, sizeof(int) * tmp_ivf[i].idstam));
		safe_call(hipMemcpy(tmp_ivf[i].ids, ivf[i].ids, sizeof(int) * ivf[i].idstam, hipMemcpyHostToDevice));
		
		ivf_mem_size += sizeof(int) * tmp_ivf[i].codes.n * tmp_ivf[i].codes.d;
		std::printf("IVF memory size up to now: %d MB\n",  ivf_mem_size / 1024 / 1024);
		safe_call(alloc((void **) &tmp_ivf[i].codes.mat, sizeof(int) * tmp_ivf[i].codes.n * tmp_ivf[i].codes.d));
		safe_call(hipMemcpy(tmp_ivf[i].codes.mat, ivf[i].codes.mat, sizeof(int) * ivf[i].codes.n * ivf[i].codes.d, hipMemcpyHostToDevice));
	}
	
	std::printf("Allocating %d MB for IVF\n",  ivf_mem_size / 1024 / 1024);
	safe_call(hipMemcpy(gpu_ivf, tmp_ivf, sizeof(ivf_t) * ivf_size, hipMemcpyHostToDevice));
	
	int* gpu_entry_map;
	std::printf("Allocating %d MB for entry map\n",  sizeof(int) * residual.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_entry_map, sizeof(int) * residual.n));
	safe_call(hipMemcpy(gpu_entry_map, entry_map, sizeof(int) * residual.n, hipMemcpyHostToDevice));
	
	int* gpu_starting_imgid;
	std::printf("Allocating %d MB for starting img id\n",  sizeof(int) * residual.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_starting_imgid, sizeof(int) * residual.n));
	safe_call(hipMemcpy(gpu_starting_imgid, starting_imgid, sizeof(int) * residual.n, hipMemcpyHostToDevice));
	
	//query_id_t* gpu_elements;
	//safe_call(hipMalloc((void **) &gpu_elements, sizeof(query_id_t) * residual.n)); //TODO: I dont know if this is truly needed
	//safe_call(hipMemcpy(gpu_elements, elements, sizeof(query_id_t) * residual.n, hipMemcpyHostToDevice));// TODO: need to rethink this
	
	matI gpu_idxs = idxs;
	
	std::printf("Allocating %d MB for idxs\n", sizeof(int) * idxs.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_idxs.mat, sizeof(int) * idxs.n));
	
	mat gpu_dists = dists;
	std::printf("Allocating %d MB for dists\n", sizeof(float) * dists.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_dists.mat, sizeof(float) * dists.n));

	//allocating the input buffer
	int* gpu_starting_inputid;
	std::printf("Allocating %d MB for input buffer\n", sizeof(int) * (residual.n + 1) / 1024 / 1024);
	safe_call(alloc((void **) &gpu_starting_inputid, sizeof(int) * (residual.n + 1)));
	safe_call(hipMemcpy(gpu_starting_inputid, starting_inputid, sizeof(int) * (residual.n + 1), hipMemcpyHostToDevice));
	
	Img* gpu_input;
	std::printf("Number of images: %d\n", num_imgs);
	std::printf("Image: %d\n", sizeof(Img));
	std::printf("Allocating %d MB for images\n",  sizeof(Img) * num_imgs / 1024 / 1024);
	safe_call(alloc((void **) &gpu_input, sizeof(Img) * num_imgs));

	dim3 block(1024, 1, 1);
	dim3 grid(residual.n, 1, 1);
	
	//find biggest ivf entry
	int biggest = 0;
	for (int i = 0; i < ivf_size; i++ ) if (ivf[i].idstam > biggest) biggest = ivf[i].idstam; 

	int sm_size = 48 << 10;
	
	std::printf("Trying to allocate: %dKB in shared memory\n", PQ.ks * PQ.nsq * sizeof(float) / 1024);
	
	compute_dists<<<grid, block,  sm_size>>>(gpu_PQ, gpu_residual, gpu_ivf, gpu_entry_map, gpu_starting_imgid, gpu_starting_inputid, gpu_input, gpu_idxs, gpu_dists, k);  
	
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch compute_dists kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else std::printf("SUCESSS!\n");
	
	std::printf("After calling the kernel\n");
	
	//exit(0);
	
	//RECEIVING DATA FROM GPU
	//safe_call(hipMemcpy(elements, 0, sizeof(query_id_t) * residual.n, hipMemcpyDeviceToHost));
	
	safe_call(hipMemcpy(idxs.mat, gpu_idxs.mat , sizeof(int) * idxs.n, hipMemcpyDeviceToHost));
	safe_call(hipMemcpy(dists.mat, gpu_dists.mat, sizeof(float) * dists.n, hipMemcpyDeviceToHost));
	
	//FREEING MEMORY
	hipFree(gpu_PQ.centroids);
	hipFree(gpu_residual.mat);
	hipFree(gpu_ivf);
	
	for (int i = 0; i < ivf_size; i++) {
		hipFree(tmp_ivf[i].ids);
		hipFree(tmp_ivf[i].codes.mat);
	}
	
	hipFree(gpu_entry_map);
	hipFree(gpu_starting_imgid);
	hipFree(gpu_idxs.mat);
	hipFree(gpu_dists.mat);
	hipFree(gpu_starting_inputid);
	hipFree(gpu_input);
	
	delete[] tmp_ivf;

}
