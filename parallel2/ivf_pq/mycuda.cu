#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "mycuda.h"

#include "hip/hip_runtime_api.h"

#include <cstdio>

#define safe_call(call) if (hipSuccess != call) { err = hipGetLastError(); \
													fprintf(stderr, "Failed: call (error code %s)!\n", \
															hipGetErrorString(err)); \
													exit(EXIT_FAILURE); }

//TODO: remember to not execute queries that dont correspond to an entry on the problem
__global__ void compute_dists(pqtipo PQ, mat residual, ivf_t* ivf, int* entry_map, int* starting_imgid, query_id_t* elements, matI idxs, mat dists) {
	int d = threadIdx.x; 
	int k = threadIdx.y;
	int tid = d * PQ.ks + k;
	int qid = blockIdx.x;
	
	extern __shared__ float distab[];
	
	float* centroid = PQ.centroids + (d * PQ.ks + k) * PQ.ds;
	float* sub_residual = residual.mat + qid * PQ.nsq * PQ.ds + d * PQ.ds;
	float dist = 0;
	
	for (int i = 0; i < PQ.ds; i++) {
		float diff = sub_residual[i] - centroid[i];
		dist += diff * diff;
	}
	
	distab[PQ.ks * d + k] = dist;

	if (tid < residual.n) { //TODO: its very likely that this is unneeded
		ivf_t entry = ivf[entry_map[qid]];
		
		if (threadIdx.x == 0 && threadIdx.y == 0) { //only one thread per block should do this, since they all refer to the same query
			atomicAdd(&elements[qid].tam, entry.idstam); //atomic because we will have up to w threads trying to increase this at the same time
		}
		
		int block_size = blockDim.x * blockDim.y;

		for (int i = tid; i < entry.idstam; i += block_size) {
			float dist = 0;

			for (int s = 0; s < PQ.nsq; s++) {
				dist += distab[PQ.ks * s + entry.codes.mat[PQ.nsq * i + s]];
			}

			dists.mat[starting_imgid[qid] + i] = dist;
			idxs.mat[starting_imgid[qid] + i] = entry.ids[i];
		}
	}	
}

void core_gpu(pqtipo PQ, mat residual, ivf_t* ivf, int ivf_size, int* entry_map, int* starting_imgid, query_id_t* elements, matI idxs, mat dists) {
	//TODO: implement / redo the error handling so that we have less code duplication
	hipError_t err = hipSuccess;
	
	pqtipo gpu_PQ = PQ;
	safe_call(hipMalloc((void **) &gpu_PQ.centroids, sizeof(float) * PQ.centroidsd * PQ.centroidsn));
	safe_call(hipMemcpy(gpu_PQ.centroids, PQ.centroids, sizeof(float) * PQ.centroidsd * PQ.centroidsn, hipMemcpyHostToDevice));
	
	mat gpu_residual = residual;
	safe_call(hipMalloc((void **) &gpu_residual.mat, sizeof(float) * residual.n * residual.d));
	safe_call(hipMemcpy(gpu_residual.mat, residual.mat, sizeof(float) * residual.n * residual.d, hipMemcpyHostToDevice));
	
	std::printf("residual.n=%d and residual.d=%d\n", residual.n, residual.d);
	
	ivf_t* gpu_ivf;
	safe_call(hipMalloc((void **) &gpu_ivf, sizeof(ivf_t) * ivf_size));
	
	ivf_t* tmp_ivf = new ivf_t[ivf_size];
	
	for (int i = 0; i < ivf_size; i++) {
		tmp_ivf[i].idstam = ivf[i].idstam; 
		tmp_ivf[i].codes = ivf[i].codes;
		
		safe_call(hipMalloc((void **) &tmp_ivf[i].ids, sizeof(int) * tmp_ivf[i].idstam));
		safe_call(hipMemcpy(tmp_ivf[i].ids, ivf[i].ids, sizeof(int) * ivf[i].idstam, hipMemcpyHostToDevice));
		
		safe_call(hipMalloc((void **) &tmp_ivf[i].codes.mat, sizeof(int) * tmp_ivf[i].codes.n * tmp_ivf[i].codes.d));
		safe_call(hipMemcpy(tmp_ivf[i].codes.mat, ivf[i].codes.mat, sizeof(int) * ivf[i].codes.n * ivf[i].codes.d, hipMemcpyHostToDevice));
	}
	
	safe_call(hipMemcpy(gpu_ivf, tmp_ivf, sizeof(ivf_t) * ivf_size, hipMemcpyHostToDevice));
	
	int* gpu_entry_map;
	safe_call(hipMalloc((void **) &gpu_entry_map, sizeof(int) * residual.n));
	safe_call(hipMemcpy(gpu_entry_map, entry_map, sizeof(int) * residual.n, hipMemcpyHostToDevice));
	
	int* gpu_starting_imgid;
	safe_call(hipMalloc((void **) &gpu_starting_imgid, sizeof(int) * residual.n));
	safe_call(hipMemcpy(gpu_starting_imgid, starting_imgid, sizeof(int) * residual.n, hipMemcpyHostToDevice));
	
	query_id_t* gpu_elements;
	safe_call(hipMalloc((void **) &gpu_elements, sizeof(query_id_t) * residual.n)); //TODO: I dont know if this is truly needed
	safe_call(hipMemcpy(gpu_elements, elements, sizeof(query_id_t) * residual.n, hipMemcpyHostToDevice));// TODO: need to rethink this
	
	matI gpu_idxs = idxs;
	safe_call(hipMalloc((void **) &gpu_idxs.mat, sizeof(int) * idxs.n));
	
	mat gpu_dists = dists;
	safe_call(hipMalloc((void **) &gpu_dists.mat, sizeof(float) * dists.n));

	dim3 block(PQ.nsq, PQ.ks, 1);
	dim3 grid(residual.n, 1, 1);
	
	std::printf("Before calling the kernel\n");
	compute_dists<<<grid, block, sizeof(float) * PQ.ks * PQ.nsq>>>(gpu_PQ, gpu_residual, gpu_ivf, gpu_entry_map, gpu_starting_imgid, gpu_elements, gpu_idxs, gpu_dists); 
	
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch compute_dists kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else std::printf("SUCESSS!\n");
	
	std::printf("After calling the kernel\n");
	
	//RECEIVING DATA FROM GPU
	safe_call(hipMemcpy(elements, gpu_elements, sizeof(query_id_t) * residual.n, hipMemcpyDeviceToHost));
	
	safe_call(hipMemcpy(idxs.mat, gpu_idxs.mat , sizeof(int) * idxs.n, hipMemcpyDeviceToHost));
	
	safe_call(hipMemcpy(dists.mat, gpu_dists.mat, sizeof(int) * dists.n, hipMemcpyDeviceToHost));
	
	//FREEING MEMORY
	hipFree(gpu_PQ.centroids);
	hipFree(gpu_residual.mat);
	hipFree(gpu_ivf);
	
	for (int i = 0; i < ivf_size; i++) {
		hipFree(tmp_ivf[i].ids);
		hipFree(tmp_ivf[i].codes.mat);
	}
	
	hipFree(gpu_entry_map);
	hipFree(gpu_starting_imgid);
	hipFree(gpu_elements);
	hipFree(gpu_idxs.mat);
	hipFree(gpu_dists.mat);
	
	delete[] tmp_ivf;
	
	for (int i = 0; i < residual.n; i++) {
		std::printf("element[%d].id=%d and element[%d].tam=%d\n", i, elements[i].id, i, elements[i].tam);
	}
	
	//std::exit(0);
	std::printf("EXITING CORE_GPU\n");
}
