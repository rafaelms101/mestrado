#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "mycuda.h"
#include "topk.cu"

#include "hip/hip_runtime_api.h"

#include <cstdio>

#define safe_call(call) if (hipSuccess != call) { err = hipGetLastError(); \
													fprintf(stderr, "Failed call: %s\nError: %s\n", \
															#call, hipGetErrorString(err)); \
													exit(EXIT_FAILURE); }


extern __shared__ char shared_memory[];

#define ACTIVE_BLOCKS 1024


//TODO: make the merge of the w query results in the GPU (?)
__global__ void compute_dists(const pqtipo PQ, const mat residual, const ivf_t* const ivf,
		const int* entry_map, const int* const starting_inputid,
		Img* distance_buffer, int block_buffer_size, matI idxs, mat dists, const int k) {
	auto tid = threadIdx.x;
	auto nthreads = blockDim.x;
	auto bid = blockIdx.x;
	auto numBlocks = gridDim.x;

	float* distab = (float*) shared_memory;

	for (int qid = bid; qid < residual.n; qid += numBlocks) {
		//computing disttab
		float* current_residual = residual.mat + qid * PQ.nsq * PQ.ds;
		int step_size = (PQ.ks * PQ.nsq + nthreads - 1) / nthreads;

		int begin_i = tid * step_size;
		int end_i = min(begin_i + step_size, PQ.ks * PQ.nsq) - 1;

		float* centroid = PQ.centroids + begin_i * PQ.ds;

		for (int i = begin_i; i <= end_i; i++) {
			int d = i / PQ.ks;

			float* sub_residual = current_residual + d * PQ.ds;
			float dist = 0;


			for (int j = 0; j < PQ.ds; j++, centroid++) {
				float diff = sub_residual[j] - *centroid;
				dist += diff * diff;
			}

			distab[i] = dist;
		}

		__syncthreads();

		//computing the distances to the vectors
		ivf_t entry = ivf[entry_map[qid]];
		Img* block_input = distance_buffer + bid * block_buffer_size; //+ starting_inputid[qid];

		for (int i = tid; i < entry.idstam; i += nthreads) {
			float dist = 0;

			for (int s = 0; s < PQ.nsq; s++) {
				dist += distab[PQ.ks * s + entry.codes.mat[PQ.nsq * i + s]];
			}

			block_input[i] = {dist, entry.ids[i]};
		}

		__syncthreads();
//		//choosing the top k


		//TODO: remember to analyze the case where size < k or size < 2k
		// selecting num_heaps
		auto shared_memory_size = 48 << 10; //TODO: there might be some function to obtain the shared memory size from the environment
		auto heap_size = k * sizeof(Entry<Img>);
		auto max_heaps = shared_memory_size / heap_size;
		auto num_subheaps = max_heaps - 1;

		if (num_subheaps > blockDim.x) num_subheaps = blockDim.x;

		if (num_subheaps * 2 * k > entry.idstam) {
			num_subheaps = entry.idstam / (2 * k);
		}

		if (num_subheaps == 0) num_subheaps = 1;

		topk(qid, num_subheaps, k, block_input, starting_inputid, dists.mat,
				idxs.mat);

		__syncthreads();
	}
}

hipError_t alloc(void **devPtr, size_t size) {
	return hipMalloc(devPtr, size);
}



void core_gpu(pqtipo PQ, mat residual, ivf_t* ivf, int ivf_size, int* entry_map, int* starting_imgid,  int* starting_inputid,  int num_imgs, matI idxs, mat dists, int k) {

	//TODO: implement / redo the error handling so that we have less code duplication
	hipError_t err = hipSuccess;

	pqtipo gpu_PQ = PQ;

	std::printf("Allocating %d MB for centroids\n",  sizeof(float) * PQ.centroidsd * PQ.centroidsn / 1024 / 1024);
	safe_call(alloc((void **) &gpu_PQ.centroids, sizeof(float) * PQ.centroidsd * PQ.centroidsn));
	safe_call(hipMemcpy(gpu_PQ.centroids, PQ.centroids, sizeof(float) * PQ.centroidsd * PQ.centroidsn, hipMemcpyHostToDevice));

	mat gpu_residual = residual;
	std::printf("Allocating %d MB for residuals\n",  sizeof(float) * residual.n * residual.d / 1024 / 1024);
	safe_call(alloc((void **) &gpu_residual.mat, sizeof(float) * residual.n * residual.d));
	safe_call(hipMemcpy(gpu_residual.mat, residual.mat, sizeof(float) * residual.n * residual.d, hipMemcpyHostToDevice));


	long ivf_mem_size = 0;
	ivf_t* gpu_ivf;

	ivf_mem_size += sizeof(ivf_t) * ivf_size;
	std::printf("IVF memory size up to now: %d MB\n",  ivf_mem_size / 1024 / 1024);
	safe_call(alloc((void **) &gpu_ivf, sizeof(ivf_t) * ivf_size));


	ivf_t* tmp_ivf = new ivf_t[ivf_size];
	int biggest_idstam = 0;

	for (int i = 0; i < ivf_size; i++) {
		if (ivf[i].idstam > biggest_idstam) biggest_idstam = ivf[i].idstam;

		tmp_ivf[i].idstam = ivf[i].idstam;
		tmp_ivf[i].codes = ivf[i].codes;

		ivf_mem_size += sizeof(int) * tmp_ivf[i].idstam;
		std::printf("IVF memory size up to now: %d MB\n",  ivf_mem_size / 1024 / 1024);
		safe_call(alloc((void **) &tmp_ivf[i].ids, sizeof(int) * tmp_ivf[i].idstam));
		safe_call(hipMemcpy(tmp_ivf[i].ids, ivf[i].ids, sizeof(int) * ivf[i].idstam, hipMemcpyHostToDevice));

		ivf_mem_size += sizeof(int) * tmp_ivf[i].codes.n * tmp_ivf[i].codes.d;
		std::printf("IVF memory size up to now: %d MB\n",  ivf_mem_size / 1024 / 1024);
		safe_call(alloc((void **) &tmp_ivf[i].codes.mat, sizeof(int) * tmp_ivf[i].codes.n * tmp_ivf[i].codes.d));
		std::printf("entry=%d, idstam=%d, codes.n=%d, codes.d=%d\n", i, tmp_ivf[i].idstam, tmp_ivf[i].codes.n, tmp_ivf[i].codes.d);
		safe_call(hipMemcpy(tmp_ivf[i].codes.mat, ivf[i].codes.mat, sizeof(int) * ivf[i].codes.n * ivf[i].codes.d, hipMemcpyHostToDevice));
	}

	std::printf("Allocating %d MB for IVF\n",  ivf_mem_size / 1024 / 1024);
	safe_call(hipMemcpy(gpu_ivf, tmp_ivf, sizeof(ivf_t) * ivf_size, hipMemcpyHostToDevice));

	int* gpu_entry_map;
	std::printf("Allocating %d MB for entry map\n",  sizeof(int) * residual.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_entry_map, sizeof(int) * residual.n));
	safe_call(hipMemcpy(gpu_entry_map, entry_map, sizeof(int) * residual.n, hipMemcpyHostToDevice));


	matI gpu_idxs = idxs;

	std::printf("Allocating %d MB for idxs\n", sizeof(int) * idxs.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_idxs.mat, sizeof(int) * idxs.n));

	mat gpu_dists = dists;
	std::printf("Allocating %d MB for dists\n", sizeof(float) * dists.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_dists.mat, sizeof(float) * dists.n));

	//allocating the input buffer
	int* gpu_starting_inputid;
	std::printf("Allocating %d MB for input buffer\n", sizeof(int) * (residual.n + 1) / 1024 / 1024);
	safe_call(alloc((void **) &gpu_starting_inputid, sizeof(int) * (residual.n + 1)));
	safe_call(hipMemcpy(gpu_starting_inputid, starting_inputid, sizeof(int) * (residual.n + 1), hipMemcpyHostToDevice));

	std::printf("Allocating %d MB for the distance buffer\n",  sizeof(Img) * biggest_idstam * ACTIVE_BLOCKS / 1024 / 1024);
	Img* gpu_distance_buffer;
	safe_call(alloc((void **) &gpu_distance_buffer, sizeof(Img) * biggest_idstam * ACTIVE_BLOCKS));


	dim3 block(1024, 1, 1);
	dim3 grid(ACTIVE_BLOCKS, 1, 1);

	//find biggest ivf entry
	int biggest = 0;
	for (int i = 0; i < ivf_size; i++ ) if (ivf[i].idstam > biggest) biggest = ivf[i].idstam;

	int sm_size = 48 << 10;

	std::printf("Trying to allocate: %dKB in shared memory\n", 48 << 10 / 1024);
	std::printf("distab needs: %dKB in shared memory\n",  PQ.ks * PQ.nsq * sizeof(float) / 1024);

	compute_dists<<<grid, block,  sm_size>>>(gpu_PQ, gpu_residual, gpu_ivf, gpu_entry_map, gpu_starting_inputid, gpu_distance_buffer, biggest_idstam, gpu_idxs, gpu_dists, k);

	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch compute_dists kernel.\nError: %s\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else std::printf("SUCESSS!\n");

	std::printf("After calling the kernel\n");


	safe_call(hipMemcpy(idxs.mat, gpu_idxs.mat , sizeof(int) * idxs.n, hipMemcpyDeviceToHost));
	safe_call(hipMemcpy(dists.mat, gpu_dists.mat, sizeof(float) * dists.n, hipMemcpyDeviceToHost));

	//FREEING MEMORY
	hipFree(gpu_PQ.centroids);
	hipFree(gpu_residual.mat);
	hipFree(gpu_ivf);

	for (int i = 0; i < ivf_size; i++) {
		hipFree(tmp_ivf[i].ids);
		hipFree(tmp_ivf[i].codes.mat);
	}

	hipFree(gpu_entry_map);
	hipFree(gpu_idxs.mat);
	hipFree(gpu_dists.mat);
	hipFree(gpu_starting_inputid);
	hipFree(gpu_distance_buffer);

	delete[] tmp_ivf;

}
