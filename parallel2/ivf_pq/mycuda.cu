#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "debug.h"
#include "mycuda.h"
#include "topk.cu"

#include "hip/hip_runtime_api.h"

#include <cstdio>

#define safe_call(call) if (hipSuccess != call) { err = hipGetLastError(); \
													fprintf(stderr, "Failed call: %s\nError: %s\n", \
															#call, hipGetErrorString(err)); \
													exit(EXIT_FAILURE); }


extern __shared__ char shared_memory[];

#define ACTIVE_BLOCKS 32
//
static pqtipo gpu_PQ;
static ivf_t* gpu_ivf;


hipError_t err = hipSuccess; //TODO: find a way to not have a global

//TODO: preload PQ.centroids and ivf
__global__ void compute_dists(const pqtipo PQ, ivf_t* ivf, const mat residual, 
		const int* rid_to_ivf, const int* const qid_to_starting_outid,
		Img* distance_buffer, int block_buffer_size, matI idxs, mat dists, const int k, const int w) {
	auto tid = threadIdx.x;
	auto nthreads = blockDim.x;
	auto bid = blockIdx.x;
	auto numBlocks = gridDim.x;
	
	float* distab = (float*) shared_memory;
	
	for (int qid = bid; qid < residual.n / w; qid += numBlocks) {
		Img* block_input = distance_buffer + bid * block_buffer_size;
		Img* current_block_input = block_input;
		int numDists = 0;
		
		for (int current_w = 0; current_w < w; current_w++) {
			//computing disttab
			int rid = qid * w + current_w;
			float* current_residual = residual.mat + rid * PQ.nsq * PQ.ds;

			int step_size = (PQ.ks * PQ.nsq + nthreads - 1) / nthreads;

			int begin_i = tid * step_size;
			int end_i = min(begin_i + step_size, PQ.ks * PQ.nsq) - 1;


			for (int i = begin_i; i <= end_i; i++) {
				float* centroid = PQ.centroids + i * PQ.ds;
				int d = i / PQ.ks;

				float* sub_residual = current_residual + d * PQ.ds;
				float dist = 0;

				for (int j = 0; j < PQ.ds; j++) {
					float diff = sub_residual[j] - centroid[j];
					dist += diff * diff;
				}

				distab[i] = dist;
			}

			__syncthreads();

			//computing the distances to the vectors
			ivf_t entry = ivf[rid_to_ivf[rid]];
			

			for (int i = tid; i < entry.idstam; i += nthreads) {
				float dist = 0;

				for (int s = 0; s < PQ.nsq; s++) {
					dist += distab[PQ.ks * s + entry.codes.mat[PQ.nsq * i + s]];
				}

				current_block_input[i].dist = dist;
				current_block_input[i].imgid = entry.ids[i];
			}
			
			current_block_input = current_block_input + entry.idstam;
			numDists += entry.idstam;
			
			__syncthreads();
		}
		

		//choosing the top k
		// selecting num_heaps
		auto shared_memory_size = 48 << 10; //TODO: there might be some function to obtain the shared memory size from the environment
		auto heap_size = k * sizeof(Entry<Img>);
		auto max_heaps = shared_memory_size / heap_size;
		auto num_subheaps = max_heaps - 1;

		if (num_subheaps > blockDim.x) num_subheaps = blockDim.x;

		if (num_subheaps * 2 * k > numDists) {
			num_subheaps = numDists / (2 * k);
		}

		if (num_subheaps == 0) num_subheaps = 1;

		topk(qid, num_subheaps, k, block_input, numDists, qid_to_starting_outid, dists.mat, idxs.mat);

		__syncthreads();
	}
}

hipError_t alloc(void **devPtr, size_t size) {
	return hipMalloc(devPtr, size);
}


void core_gpu(pqtipo PQ, mat residual, ivf_t* ivf, int ivf_size, int* rid_to_ivf, int* qid_to_starting_outid, matI idxs, mat dists, int k, int w) {
	//TODO: implement / redo the error handling so that we have less code duplication
	mat gpu_residual = residual;
	debug("Allocating %d MB for residuals\n",  sizeof(float) * residual.n * residual.d / 1024 / 1024);
	safe_call(alloc((void **) &gpu_residual.mat, sizeof(float) * residual.n * residual.d));
	safe_call(hipMemcpy(gpu_residual.mat, residual.mat, sizeof(float) * residual.n * residual.d, hipMemcpyHostToDevice));


	int biggest_idstam = 0;
	for (int i = 0; i < ivf_size; i++) {
		if (ivf[i].idstam > biggest_idstam) biggest_idstam = ivf[i].idstam;
	}

	int* gpu_rid_to_ivf;
	debug("Allocating %d MB for rid_to_ivf\n",  sizeof(int) * residual.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_rid_to_ivf, sizeof(int) * residual.n));
	safe_call(hipMemcpy(gpu_rid_to_ivf, rid_to_ivf, sizeof(int) * residual.n, hipMemcpyHostToDevice));


	matI gpu_idxs = idxs;
	debug("Allocating %d MB for idxs\n", sizeof(int) * idxs.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_idxs.mat, sizeof(int) * idxs.n));

	mat gpu_dists = dists;
	debug("Allocating %d MB for dists\n", sizeof(float) * dists.n / 1024 / 1024);
	safe_call(alloc((void **) &gpu_dists.mat, sizeof(float) * dists.n));

	//allocating the input buffer
	int* gpu_qid_to_starting_outid;
	debug("Allocating %d MB for qid_to_starting_outid\n", sizeof(int) * residual.n / w / 1024 / 1024);
	safe_call(alloc((void **) &gpu_qid_to_starting_outid, sizeof(int) * residual.n / w));
	safe_call(hipMemcpy(gpu_qid_to_starting_outid, qid_to_starting_outid, sizeof(int) * residual.n / w, hipMemcpyHostToDevice));

	debug("Allocating %d MB for the distance buffer\n",  sizeof(Img) * biggest_idstam * w * ACTIVE_BLOCKS / 1024 / 1024);
	Img* gpu_distance_buffer;
	safe_call(alloc((void **) &gpu_distance_buffer, sizeof(Img) * biggest_idstam * w * ACTIVE_BLOCKS)); //TODO: its possible to save some memory if I compute the biggest AGGREGATED idstam (as in, the sum of all w idstam)


	dim3 block(1024, 1, 1);
	dim3 grid(ACTIVE_BLOCKS, 1, 1);

	int sm_size = 48 << 10;

	debug("Trying to allocate: %dKB in shared memory\n", 48 << 10 / 1024);
	debug("distab needs: %dKB in shared memory\n",  PQ.ks * PQ.nsq * sizeof(float) / 1024);

	compute_dists<<<grid, block,  sm_size>>>(gpu_PQ, gpu_ivf, gpu_residual, gpu_rid_to_ivf, gpu_qid_to_starting_outid, gpu_distance_buffer, biggest_idstam * w, gpu_idxs, gpu_dists, k, w);

	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch compute_dists kernel.\nError: %s\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} 

	debug("After calling the kernel\n");


	safe_call(hipMemcpy(idxs.mat, gpu_idxs.mat , sizeof(int) * idxs.n, hipMemcpyDeviceToHost));
	safe_call(hipMemcpy(dists.mat, gpu_dists.mat, sizeof(float) * dists.n, hipMemcpyDeviceToHost));

	//FREEING MEMORY
	hipFree(gpu_residual.mat);
	hipFree(gpu_rid_to_ivf);
	hipFree(gpu_idxs.mat);
	hipFree(gpu_dists.mat);
	hipFree(gpu_qid_to_starting_outid);
	hipFree(gpu_distance_buffer);
}

ivf_t* tmp_ivf;
int tmp_size;

void preallocate_gpu_mem(pqtipo host_PQ, ivf_t* host_ivf, int ivf_size) {
	long ivf_mem_size = 0;
	safe_call(alloc((void **) &gpu_ivf, sizeof(ivf_t) * ivf_size));	
	
	ivf_mem_size += sizeof(ivf_t) * ivf_size;
	tmp_ivf = new ivf_t[ivf_size];
	tmp_size = ivf_size;
	
	for (int i = 0; i < ivf_size; i++) {
		tmp_ivf[i].idstam = host_ivf[i].idstam;
		tmp_ivf[i].codes = host_ivf[i].codes;

		ivf_mem_size += sizeof(int) * tmp_ivf[i].idstam;
		safe_call(alloc((void ** ) &tmp_ivf[i].ids, sizeof(int) * tmp_ivf[i].idstam));
		safe_call(hipMemcpy(tmp_ivf[i].ids, host_ivf[i].ids, sizeof(int) * host_ivf[i].idstam, hipMemcpyHostToDevice));
		ivf_mem_size += sizeof(int) * tmp_ivf[i].codes.n * tmp_ivf[i].codes.d;
		safe_call(alloc((void ** ) &tmp_ivf[i].codes.mat, sizeof(int) * tmp_ivf[i].codes.n * tmp_ivf[i].codes.d));
		safe_call(hipMemcpy(tmp_ivf[i].codes.mat, host_ivf[i].codes.mat, sizeof(int) * host_ivf[i].codes.n * host_ivf[i].codes.d, hipMemcpyHostToDevice));
	}

	debug("Allocating %d MB for IVF\n",  ivf_mem_size / 1024 / 1024);
	safe_call(hipMemcpy(gpu_ivf, tmp_ivf, sizeof(ivf_t) * ivf_size, hipMemcpyHostToDevice));
	
	//centroids
	gpu_PQ = host_PQ;
	debug("Allocating %d MB for centroids\n",  sizeof(float) * host_PQ.centroidsd * host_PQ.centroidsn / 1024 / 1024);
	safe_call(alloc((void **) &gpu_PQ.centroids, sizeof(float) * host_PQ.centroidsd * host_PQ.centroidsn));
	safe_call(hipMemcpy(gpu_PQ.centroids, host_PQ.centroids, sizeof(float) * host_PQ.centroidsd * host_PQ.centroidsn, hipMemcpyHostToDevice));
}

void deallocate_gpu_mem() {
	for (int i = 0; i < tmp_size; i++) {
		hipFree(tmp_ivf[i].ids);
		hipFree(tmp_ivf[i].codes.mat);
		
	}
	
	hipFree(gpu_ivf);
	hipFree(gpu_PQ.centroids);
	
	delete tmp_ivf;
}
