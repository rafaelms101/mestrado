#include <hip/hip_runtime.h>
#include <cstdio>
#include <utility>

#include <cmath>
#include <vector>

#include <cstdio>
#include <utility>

enum class HeapType {
	kMinHeap, kMaxHeap
};
enum class PreferIndices {
	kLower, kHigher
};

struct Img {
	float dist;
	int imgid;

	__device__ bool operator==(const Img& r) const {
		return r.imgid == imgid;
	}

	__device__ bool operator<(const Img& r) const {
		return dist > r.dist || (dist == r.dist && imgid > r.imgid);
	}

	__device__ bool operator>(const Img& r) const {
		return dist < r.dist || (dist == r.dist && imgid < r.imgid);
	}
};

template<typename T>
struct Entry {
	int index;
	T value;

	// Test-only.
	static bool greater(const Entry<T>& a, const Entry<T>& b) {
		if (a.value == b.value) {
			return a.index < b.index;
		}
		return a.value > b.value;
	}
};

template<typename T>
struct LinearData {
	typedef Entry<T> Entry;

	__device__ Entry& operator[](std::size_t index) const {
		return data[index];
	}

	__device__ int get_index(int i) const {
		return data[i].index;
	}
	__device__ T get_value(int i) const {
		return data[i].value;
	}

	Entry* const data;
};

template<typename T>
struct IndirectLinearData {
	typedef Entry<T> Entry;

	__device__ Entry& operator[](std::size_t index) const {
		return data[index];
	}

	__device__ int get_index(int i) const {
		return backing_data[data[i].index].index;
	}
	__device__ T get_value(int i) const {
		return data[i].value;
	}

	Entry* const data;
	Entry* const backing_data;
};

template<typename T>
struct StridedData {
	typedef Entry<T> Entry;

	__device__ Entry& operator[](std::size_t index) const {
		return data[index * num_subheaps + threadIdx.x];
	}

	__device__ int get_index(int i) const {
		return (*this)[i].index;
	}
	
	__device__ T get_value(int i) const {
		return (*this)[i].value;
	}

	Entry* const data;
	int num_subheaps;
};

// A heap of Entry<T> that can either work as a min-heap or as a max-heap.
template<HeapType heapType, PreferIndices preferIndices,
		template<typename > class Data, typename T>
struct IndexedHeap {
	typedef typename Data<T>::Entry Entry;
	const Data<T> data;

	__device__ bool is_above(int left, int right) {
		T left_value = data.get_value(left);
		T right_value = data.get_value(right);
		if (left_value == right_value) {
			if (preferIndices == PreferIndices::kLower) {
				return data.get_index(left) < data.get_index(right);
			} else {
				return data.get_index(left) > data.get_index(right);
			}
		}
		if (heapType == HeapType::kMinHeap) {
			return left_value < right_value;
		} else {
			return left_value > right_value;
		}
	}

	__device__ void assign(int i, const Entry& entry) {
		data[i] = entry;
	}

	__device__ void push_up(int i) {
		int child = i;
		int parent;
		for (; child > 0; child = parent) {
			parent = (child - 1) / 2;
			if (!is_above(child, parent)) {
				// Heap property satisfied.
				break;
			}
			swap(child, parent);
		}
	}

	__device__ void swap(int a, int b) {
		auto tmp = data[b];
		data[b] = data[a];
		data[a] = tmp;
	}

	__device__ void push_root_down(int k) {
		push_down(0, k);
	}

	// MAX-HEAPIFY in Cormen
	__device__ void push_down(int node, int k) {
		while (true) {
			const int left = 2 * node + 1;
			const int right = left + 1;
			int smallest = node;
			if (left < k && is_above(left, smallest)) {
				smallest = left;
			}
			if (right < k && is_above(right, smallest)) {
				smallest = right;
			}
			if (smallest == node) {
				break;
			}
			swap(smallest, node);
			node = smallest;
		}
	}

	// BUILD-MAX-HEAPIFY in Cormen
	__device__ void build(int k) {
		for (int node = (k - 1) / 2; node >= 0; node--) {
			push_down(node, k);
		}
	}

	// HEAP-EXTRACT-MAX in Cormen
	__device__ void remove_root(int k) {
		data[0] = data[k - 1];
		push_root_down(k - 1);
	}

	// in-place HEAPSORT in Cormen
	// This method destroys the heap property.
	__device__ void sort(int k) {
		for (int slot = k - 1; slot > 0; slot--) {
			// This is like remove_root but we insert the element at the end.
			swap(slot, 0);
			// Heap is now an element smaller.
			push_root_down(/*k=*/slot);
		}
	}

	__device__ void replace_root(const Entry& entry, int k) {
		data[0] = entry;
		push_root_down(k);
	}

	__device__ const Entry& root() {
		return data[0];
	}
};

template<HeapType heapType, PreferIndices preferIndices,
		template<typename > class Data, typename T>
__device__ IndexedHeap<heapType, preferIndices, Data, T> make_indexed_heap(
		typename Data<T>::Entry* data, int num_shards) {
	return IndexedHeap<heapType, preferIndices, Data, T> { Data<T> { data, num_shards } };
}

// heapTopK walks over [input, input+length) with `step_size` stride starting at
// `start_index`.
// It builds a top-`k` heap that is stored in `heap_entries` using `Accessor` to
// access elements in `heap_entries`. If sorted=true, the elements will be
// sorted at the end.
template<typename T, template<typename > class Data = LinearData>
__device__ void heapTopK(const T* __restrict__ block_input, int length, int k,
		Entry<T>* __restrict__ shared, int num_subheaps, bool sorted = false,
		int start_index = 0, int step_size = 1) {

	auto heap = make_indexed_heap<HeapType::kMinHeap, PreferIndices::kHigher,
			Data, T>(shared, num_subheaps);

	int heap_end_index = start_index + k * step_size;
	if (heap_end_index > length) {
		heap_end_index = length;
	}
	// Initialize the min-heap.
	int slot = 0;
	for (int index = start_index; index < heap_end_index; index += step_size, slot++) {
		heap.assign(slot, { index, block_input[index] });
	}

	heap.build(slot); //TODO: [before it was heap.build(k)] verify if the heap building function works when you havent assigned all the elements

	// Now iterate over the remaining items.
	// If an item is smaller than the min element, it is not amongst the top k.
	// Otherwise, replace the min element with it and push upwards.
	for (int index = heap_end_index; index < length; index += step_size) {
		// We prefer elements with lower indices. This is given here.
		// Later elements automatically have higher indices, so can be discarded.
		if (block_input[index] > heap.root().value) {
			// This element should replace the min.
			heap.replace_root( { index, block_input[index] }, k);
		}
	}

	// Sort if wanted.
	if (sorted) {
		heap.sort(k);
	}
}

// mergeShards performs a top-k merge on `num_shards` many sorted streams that
// are sorted and stored in `entries` in a strided way:
// |s_1 1st|s_2 1st|...s_{num_shards} 1st|s_1 2nd|s_2 2nd|...
// The overall top k elements are written to `top_k_values` and their indices
// to top_k_indices.
// `top_k_heap` is used as temporary storage for the merge heap.
__device__ void mergeShards(int num_shards, int k,
		Entry<Img>* __restrict__ entries, Entry<Img>* __restrict__ top_k_heap,
		float* top_k_values, int* top_k_indices) {
	// If k < num_shards, we can use a min-heap with k elements to get the top k
	// of the sorted blocks.
	// If k > num_shards, we can initialize a min-heap with the top element from
	// each sorted block.
	const int heap_size = k < num_shards ? k : num_shards;

	// Min-heap part.
	{
		auto min_heap = IndexedHeap<HeapType::kMinHeap, PreferIndices::kHigher,
				IndirectLinearData, Img> { IndirectLinearData<Img> { top_k_heap,
				entries } };
		// Initialize the heap as a min-heap.
		for (int slot = 0; slot < heap_size; slot++) {
			min_heap.assign(slot, { slot, entries[slot].value });
		}
		min_heap.build(heap_size);

		// Now perform top k with the remaining shards (if num_shards > heap_size).
		for (int shard = heap_size; shard < num_shards; shard++) {
			const auto entry = entries[shard];
			const auto root = min_heap.root();
			if (entry.value < root.value) {
				continue;
			}
			if (entry.value == root.value
					&& entry.index > entries[root.index].index) {
				continue;
			}
			// This element should replace the min.
			min_heap.replace_root( { shard, entry.value }, heap_size);
		}
	}

	// Max-part.
	{
		// Turn the min-heap into a max-heap in-place.
		auto max_heap = IndexedHeap<HeapType::kMaxHeap, PreferIndices::kLower,
				IndirectLinearData, Img> { IndirectLinearData<Img> { top_k_heap,
				entries } };
		// Heapify into a max heap.
		max_heap.build(heap_size);

		// Now extract the minimum k-1 times.
		// k is treated specially.
		const int last_k = k - 1;
		for (int rank = 0; rank < last_k; rank++) {
			const Entry<Img>& max_element = max_heap.root();
			top_k_values[rank] = max_element.value.dist;

			int shard_index = max_element.index;
			top_k_indices[rank] = entries[shard_index].value.imgid;

			int next_shard_index = shard_index + num_shards;
			// For rank < k-1, each top k heap still contains at least 1 element,
			// so we can draw a replacement.
			max_heap.replace_root(
					{ next_shard_index, entries[next_shard_index].value },
					heap_size);
		}

		// rank == last_k.
		const Entry<Img>& max_element = max_heap.root();
		top_k_values[last_k] = max_element.value.dist;

		int shard_index = max_element.index;
		top_k_indices[last_k] = entries[shard_index].value.imgid;
	}
}

extern __shared__ char shared_memory[];

__device__ void TopKKernel(const int qid, const int num_subheaps, const Img* input,
		const int* const starting_inputid, const int k, const bool sorted, float* output,
		int* indices) {
	const Img* block_input = input + starting_inputid[qid];
	auto tid = threadIdx.x;

	Entry<Img>* shared = (Entry<Img>*) shared_memory;

	
	int length = starting_inputid[qid + 1] - starting_inputid[qid]; //TODO: find a better solution for passing along the number of images
	
	if (tid < num_subheaps) {
		heapTopK<Img, StridedData>(block_input, length, k, shared, num_subheaps, true, tid,  num_subheaps);
	}
	
	__syncthreads();
	
	if (tid == 0) {
		float* block_output = output + qid * k;
		int* batch_indices = indices + qid * k;
		Entry<Img>* top_k_heap = shared + num_subheaps  * k;

		// TODO(blackhc): Erich says: Performance can likely be improved
		// significantly by having the merge be done by multiple threads rather than
		// just one.  ModernGPU has some nice primitives that could help with this.
		mergeShards(num_subheaps, k, shared, top_k_heap, block_output,
				batch_indices);
	}
}

/*
 template <typename T>
 cudaError LaunchTopKKernel(const cudaStream_t& stream, int num_shards,
 const T* input, int batch_size, int length, int k,
 bool sorted, T* output, int* indices) {
 // This code assumes that k is small enough that the computation
 // fits inside shared memory (hard coded to 48KB).  In practice this
 // means k <= 3072 for T=float/int32 and k <= 2048 for T=double/int64.
 // The calculation is:
 //   shared_memory_size / (2 * (sizeof(int) + sizeof(T))) < k.

 // Use as many shards as possible.
 if (num_shards <= 0) {
 constexpr auto shared_memory_size = 48 << 10;  // 48 KB
 const auto heap_size = k * sizeof(Entry<T>);
 // shared_memory_size = (num_shards + 1) * heap_size <=>
 num_shards = shared_memory_size / heap_size - 1;
 if (num_shards <= 0) {
 num_shards = 1;
 }
 auto shard_size = length / num_shards;
 auto min_shard_size = 2 * k;
 if (shard_size < min_shard_size) {
 num_shards = length / min_shard_size;
 }
 if (num_shards <= 0) {
 num_shards = 1;
 } else if (num_shards > 1024) {
 num_shards = 1024;
 }
 }
 // We are limited by the amount of shared memory we have per block.
 auto shared_memory_size = (num_shards + 1) * k * sizeof(Entry<T>);

 TopKKernel<<<batch_size, num_shards, shared_memory_size, stream>>>(
 input, length, k, sorted, output, indices);
 return cudaGetLastError();
 }*/

__device__ void topk(const int qid, const int num_subheaps, const int k, Img* input, const int* const starting_inputid,
		float* output, int* indexes) {
		TopKKernel(qid, num_subheaps, input, starting_inputid, k, false, output,
						indexes);
}

/*
 struct Entry {
 float value;
 int imgid;
 int index;
 };

 __device__ void sort_strided(Entry* entries, int length) {
 int tid = threadIdx.x;
 int numThreads = blockDim.x;
 
 for (int new_element_id = tid + numThreads; new_element_id < length; new_element_id += numThreads) {
 //Trying to insert element with index id in the sorted array
 
 int id;
 for (id = new_element_id - numThreads; id >= 0; id -= numThreads) {
 if (entries[new_element_id].value <= entries[id].value) break;
 }
 
 Entry to_be_inserted = entries[new_element_id];
 
 //now we shift everyone right
 for (int insertion_id = id + numThreads; insertion_id <= new_element_id; insertion_id += numThreads) {
 Entry tmp = entries[insertion_id];
 entries[insertion_id] = to_be_inserted;
 to_be_inserted = tmp;
 }
 }
 }

 __device__ void sort(Entry* entries, int length) {
 for (int new_element_id = 1; new_element_id < length; new_element_id += 1) {
 //Trying to insert element with index id in the sorted array
 
 int id;
 for (id = new_element_id - 1; id >= 0; id -= 1) {
 if (entries[new_element_id].value <= entries[id].value) break;
 }
 
 Entry to_be_inserted = entries[new_element_id];
 
 //now we shift everyone right
 for (int insertion_id = id + 1; insertion_id <= new_element_id; insertion_id += 1) {
 Entry tmp = entries[insertion_id];
 entries[insertion_id] = to_be_inserted;
 to_be_inserted = tmp;
 }
 }
 }

 __device__ void insert(Entry* entries, int new_id, int k) {
 Entry new_element = entries[new_id];

 int i;

 for (i = k - 1; i >= 0; i--) {
 if (new_element.value > entries[i].value) {
 entries[i + 1] = entries[i];
 } else break;
 }

 entries[i + 1] = new_element;
 }

 //TODO: test the case where  the number of entries is too small
 __device__ void topk(int k, Entry* entry, int length) {
 int tid = threadIdx.x;
 int stride = blockDim.x;

 sort_strided(entry, length);

 for (int id = tid; id < length; id += stride) {
 entry[id].index = id;
 }
 
 __syncthreads();
 
 if (tid == 0) {
 k = min(length, k);
 int lastk = k - 1;

 sort(entry, k);
 
 int start = k;
 int end = min(k + stride - 1, length - 1);

 Entry old_smallest = entry[lastk];
 
 
 while (true) {
 for (int id = start; id <= end; id++) {
 if (entry[id].value > entry[lastk].value) {
 int next_id = entry[id].index + stride;
 insert(entry, id, k);

 if (next_id < length) {
 entry[id] = entry[next_id];
 } else {
 entry[id] = {-1, -1};
 }
 }
 }

 if (entry[lastk].value == old_smallest.value) break;
 }	
 
 }
 }
 */
